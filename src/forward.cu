#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void linear_forward(float* input, float* weight, float* bias, float* output,
                               int batch_size, int in_features, int out_features) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // output row (batch)
    int col = blockIdx.x * blockDim.x + threadIdx.x; // output col (neuron)

    if (row < batch_size && col < out_features) {
        float sum = 0.0f;
        for (int i = 0; i < in_features; ++i) {
            sum += input[row * in_features + i] * weight[col * in_features + i]; // weight is [out, in]
        }
        output[row * out_features + col] = sum + bias[col];
    }
}