#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

// ---------------------------------------------
// CUDA Kernels
// ---------------------------------------------

// Linear layer: output = input × weight^T + bias
__global__ void linear_forward(float* input, float* weight, float* bias, float* output,
                               int batch_size, int in_features, int out_features) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // which input in batch
    int col = blockIdx.x * blockDim.x + threadIdx.x; // which neuron (output feature)

    if (row < batch_size && col < out_features) {
        float sum = 0.0f;
        for (int i = 0; i < in_features; ++i) {
            sum += input[row * in_features + i] * weight[col * in_features + i];  // weight[output, input]
        }
        output[row * out_features + col] = sum + bias[col];
    }
}

// ReLU activation: element-wise max(0, x)
__global__ void relu_forward(float* input, float* output, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

// ---------------------------------------------
// Kernel Launchers (externally callable)
// ---------------------------------------------

// Wrapper to launch linear layer
void launch_linear_forward(float* input, float* weight, float* bias, float* output,
                           int batch_size, int in_features, int out_features) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((out_features + 15) / 16, (batch_size + 15) / 16);
    linear_forward<<<blocksPerGrid, threadsPerBlock>>>(input, weight, bias, output,
                                                       batch_size, in_features, out_features);
}

// Wrapper to launch ReLU
void launch_relu_forward(float* input, float* output, int size) {
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    relu_forward<<<gridSize, blockSize>>>(input, output, size);
}
