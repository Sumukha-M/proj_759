#include <iostream>          // For standard I/O operations
#include <cmath>             // For sin() and cos() functions
#include <hip/hip_runtime.h>    // CUDA runtime API

// Declare external CUDA kernel functions (definitions are in other files)
__global__ void matmul_forward(float* x, float* w, float* y);   // Forward pass kernel
__global__ void matmul_backward(float* grad_out, float* x, float* grad_w);  // Backward pass kernel
__global__ void sgd_update(float* w, float* grad_w, float lr);  // SGD weight update kernel

// Define constants for batch size (N) and feature dimension (D)
#define N 1024
#define D 512

int main() {
    // Declare pointers for input, weights, output, gradients, and learning rate
    float *x, *w, *y, *grad_out, *grad_w;
    float lr = 0.001f;   // Learning rate for SGD

    // Allocate unified memory accessible by both CPU and GPU
    hipMallocManaged(&x, N * D * sizeof(float));        // Input matrix
    hipMallocManaged(&w, D * D * sizeof(float));        // Weight matrix
    hipMallocManaged(&y, N * D * sizeof(float));        // Output matrix
    hipMallocManaged(&grad_out, N * D * sizeof(float)); // Gradient from next layer
    hipMallocManaged(&grad_w, D * D * sizeof(float));   // Gradient w.r.t weights

    // Initialize input 'x' with sine values and weights 'w' with cosine values
    for (int i = 0; i < N * D; ++i) 
        x[i] = sin(i);

    for (int i = 0; i < D * D; ++i) 
        w[i] = cos(i);

    // Define CUDA kernel launch parameters
    dim3 threads(256);             // 256 threads per block
    dim3 blocks((N + 255) / 256);  // Calculate number of blocks needed for N elements

    // Launch forward pass kernel: y = x * w
    matmul_forward<<<blocks, threads>>>(x, w, y);
    hipDeviceSynchronize();   // Wait for GPU to finish

    // Set dummy gradient output (simulate backpropagation input)
    for (int i = 0; i < N * D; ++i) 
        grad_out[i] = 1.0f;

    // Launch backward pass kernel to compute gradient w.r.t weights
    matmul_backward<<<(D + 255) / 256, threads>>>(grad_out, x, grad_w);
    hipDeviceSynchronize();   // Wait for GPU to finish

    // Launch SGD update kernel to adjust weights based on gradients
    sgd_update<<<(D * D + 255) / 256, threads>>>(w, grad_w, lr);
    hipDeviceSynchronize();   // Wait for GPU to finish

    // Print completion message
    std::cout << "One training iteration complete." << std::endl;

    // Free allocated unified memory
    hipFree(x); 
    hipFree(w); 
    hipFree(y); 
    hipFree(grad_out); 
    hipFree(grad_w);

    return 0;   // Exit program
}
