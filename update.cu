#include <hip/hip_runtime.h>

#define D 512

__global__ void sgd_update(float* w, float* grad_w, float lr) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < D * D)
        w[idx] -= lr * grad_w[idx];
}