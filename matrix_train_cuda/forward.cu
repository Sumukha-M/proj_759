#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define D 512

__global__ void matmul_forward(float* x, float* w, float* y) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N) {
        for (int j = 0; j < D; ++j) {
            y[idx * D + j] = 0;

            // Dot product: idx-th row of x and j-th column of w
            for (int k = 0; k < D; ++k) {
                y[idx * D + j] += x[idx * D + k] * w[k * D + j];
            }
        }
    }
}
