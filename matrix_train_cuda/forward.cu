#include <hip/hip_runtime.h>
#include <stdio.h>

// Define matrix dimensions
#define N 1024  // Number of rows in input matrix x (and output matrix y)
#define D 512   // Number of columns in x, rows and columns in weight matrix w

// CUDA kernel to perform matrix multiplication y = x * w
// x: [N x D], w: [D x D], y: [N x D]
__global__ void matmul_forward(float* x, float* w, float* y) {
    // Calculate the global thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Only allow threads corresponding to valid row indices of x
    if (idx < N) {
        // Iterate over each column of the output matrix y
        for (int j = 0; j < D; ++j) {
            // Initialize the output element y[idx][j] to zero
            y[idx * D + j] = 0;

            // Compute dot product between the idx-th row of x and j-th column of w
            for (int k = 0; k < D; ++k) {
                // x[idx][k] * w[k][j]
                y[idx * D + j] += x[idx * D + k] * w[k * D + j];
            }
        }
    }
}
