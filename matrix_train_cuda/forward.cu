#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define D 512

_global_ void matmul_forward(float* x, float* w, float* y) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        for (int j = 0; j < D; ++j) {
            y[idx * D + j] = 0;
            for (int k = 0; k < D; ++k)
                y[idx * D + j] += x[idx * D + k] * w[k * D + j]; //dot product of idth row of x and jth col of w
        }
    }
}
