#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define D 512

__global__ void matmul_forward(float* x, float* w, float* y) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Work only if thread index is within matrix row bounds
    if (idx < N) {
        // Future matrix computation will go here
    }
}
