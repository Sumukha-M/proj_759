#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

// Declare kernel functions from other files
__global__ void matmul_forward(float* x, float* w, float* y);
__global__ void matmul_backward(float* grad_out, float* x, float* grad_w);
__global__ void sgd_update(float* w, float* grad_w, float lr);

#define N 1024
#define D 512

int main() {
    float *x, *w, *y, *grad_out, *grad_w;
    float lr = 0.001f;
}
