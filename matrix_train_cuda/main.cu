#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void matmul_forward(float* x, float* w, float* y);
__global__ void matmul_backward(float* grad_out, float* x, float* grad_w);
__global__ void sgd_update(float* w, float* grad_w, float lr);

#define N 1024
#define D 512

int main() {
    float *x, *w, *y, *grad_out, *grad_w;
    float lr = 0.001f;

    hipMallocManaged(&x, N * D * sizeof(float));
    hipMallocManaged(&w, D * D * sizeof(float));
    hipMallocManaged(&y, N * D * sizeof(float));
    hipMallocManaged(&grad_out, N * D * sizeof(float));
    hipMallocManaged(&grad_w, D * D * sizeof(float));

    for (int i = 0; i < N * D; ++i) x[i] = sin(i);
    for (int i = 0; i < D * D; ++i) w[i] = cos(i);

    dim3 threads(256);
    dim3 blocks((N + 255) / 256);

    matmul_forward<<<blocks, threads>>>(x, w, y);
    hipDeviceSynchronize();

    for (int i = 0; i < N * D; ++i) grad_out[i] = 1.0f;

    matmul_backward<<<(D + 255) / 256, threads>>>(grad_out, x, grad_w);
    hipDeviceSynchronize();

    sgd_update<<<(D * D + 255) / 256, threads>>>(w, grad_w, lr);
    hipDeviceSynchronize();

    // Completion message and cleanup
    std::cout << "One training iteration complete." << std::endl;

    hipFree(x); 
    hipFree(w); 
    hipFree(y); 
    hipFree(grad_out); 
    hipFree(grad_w);

    return 0;
}
