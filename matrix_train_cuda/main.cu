#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void matmul_forward(float* x, float* w, float* y);
__global__ void matmul_backward(float* grad_out, float* x, float* grad_w);
__global__ void sgd_update(float* w, float* grad_w, float lr);

#define N 1024
#define D 512

int main() {
    float *x, *w, *y, *grad_out, *grad_w;
    float lr = 0.001f;

    hipMallocManaged(&x, N * D * sizeof(float));
    hipMallocManaged(&w, D * D * sizeof(float));
    hipMallocManaged(&y, N * D * sizeof(float));
    hipMallocManaged(&grad_out, N * D * sizeof(float));
    hipMallocManaged(&grad_w, D * D * sizeof(float));

    // Initialize x and w
    for (int i = 0; i < N * D; ++i) x[i] = sin(i);
    for (int i = 0; i < D * D; ++i) w[i] = cos(i);
}
