#include <hip/hip_runtime.h>

#define N 1024
#define D 512

__global__ void matmul_backward(float* grad_out, float* x, float* grad_w) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Just a placeholder loop
    if (idx < D) {
        for (int j = 0; j < D; ++j) {
            grad_w[idx * D + j] = 1.0; // Dummy value
        }
    }
}
