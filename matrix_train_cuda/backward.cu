#include <hip/hip_runtime.h>

#define N 1024
#define D 512

_global_ void matmul_backward(float* grad_out, float* x, float* grad_w) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < D) {
        for (int j = 0; j < D; ++j) {
            grad_w[idx * D + j] = 0;
            for (int b = 0; b < N; ++b)
                grad_w[idx * D + j] += x[b * D + idx] * grad_out[b * D + j];
        }
    }
}