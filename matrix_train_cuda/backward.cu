#include <hip/hip_runtime.h>

#define N 1024
#define D 512

__global__ void matmul_backward(float* grad_out, float* x, float* grad_w) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < D) {
        for (int j = 0; j < D; ++j) {
            float sum = 0;
            for (int b = 0; b < N; ++b) {
                sum += x[b * D + idx];  // Grad output not used yet
            }
            grad_w[idx * D + j] = sum;
        }
    }
}
