#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define N 1024  // Batch size (number of data samples)
#define D 512   // Feature dimension

// CUDA kernel to compute gradient of weights (grad_w) for a matrix multiplication backward pass

// This computes: grad_w = x^T * grad_out
_global_ void matmul_backward(float* grad_out, float* x, float* grad_w) {
    // Compute the global thread index (each thread handles one row of grad_w)

__global__ void matmul_backward(float* grad_out, float* x, float* grad_w) {
    // Compute the thread index in 1D

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure the thread operates only within valid feature dimensions
    if (idx < D) {
        // Loop over each column of grad_w
        for (int j = 0; j < D; ++j) {
            // Initialize the (idx, j) element of grad_w to zero before accumulation
            grad_w[idx * D + j] = 0.0f;

            // Perform the dot product of:
            // - idx-th column of input matrix x (since x^T is involved)
            // - j-th column of grad_out
            //
            // This loop runs over the batch dimension to accumulate contributions
            for (int b = 0; b < N; ++b) {
                // Accumulate: grad_w[idx][j] += x[b][idx] * grad_out[b][j]
                grad_w[idx * D + j] += x[b * D + idx] * grad_out[b * D + j];
            }
        }
    }
}
