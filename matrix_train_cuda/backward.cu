#include <hip/hip_runtime.h>

#define N 1024  // Batch size (number of data samples)
#define D 512   // Feature dimension

// CUDA kernel to compute gradient of weights (grad_w) for a matrix multiplication backward pass
__global__ void matmul_backward(float* grad_out, float* x, float* grad_w) {
    // Compute the thread index in 1D
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Each thread computes the row idx of the grad_w matrix
    if (idx < D) {
        for (int j = 0; j < D; ++j) {
            // Initialize grad_w[idx][j] to 0 before accumulation
            grad_w[idx * D + j] = 0;

            // Accumulate gradient
            for (int b = 0; b < N; ++b)
                grad_w[idx * D + j] += x[b * D + idx] * grad_out[b * D + j];
        }
    }
}
