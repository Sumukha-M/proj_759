#include <hip/hip_runtime.h>   // CUDA runtime API for kernel execution

#define D 512   // Define matrix dimension (D x D weights)

// CUDA kernel to perform SGD weight update
__global__ void sgd_update(float* w, float* grad_w, float lr) {
    // Calculate global thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure thread operates within valid range
    if (idx < D * D)
        w[idx] -= lr * grad_w[idx];   // Perform weight update: w = w - lr * grad_w
}
