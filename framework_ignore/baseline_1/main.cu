
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

#define N 1024  // batch size
#define D 512   // input/output size

__global__ void matmul_forward(float* x, float* w, float* y) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        for (int j = 0; j < D; ++j) {
            y[idx * D + j] = 0;
            for (int k = 0; k < D; ++k)
                y[idx * D + j] += x[idx * D + k] * w[k * D + j];
        }
    }
}

__global__ void matmul_backward(float* grad_out, float* x, float* grad_w) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < D) {
        for (int j = 0; j < D; ++j) {
            grad_w[idx * D + j] = 0;
            for (int b = 0; b < N; ++b)
                grad_w[idx * D + j] += x[b * D + idx] * grad_out[b * D + j];
        }
    }
}

__global__ void sgd_update(float* w, float* grad_w, float lr) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < D * D)
        w[idx] -= lr * grad_w[idx];
}

int main() {
    float *x, *w, *y, *grad_out, *grad_w;
    float lr = 0.001;

    hipMallocManaged(&x, N * D * sizeof(float));
    hipMallocManaged(&w, D * D * sizeof(float));
    hipMallocManaged(&y, N * D * sizeof(float));
    hipMallocManaged(&grad_out, N * D * sizeof(float));
    hipMallocManaged(&grad_w, D * D * sizeof(float));

    for (int i = 0; i < N * D; ++i) x[i] = sin(i);
    for (int i = 0; i < D * D; ++i) w[i] = cos(i);

    dim3 threads(256);
    dim3 blocks((N + 255) / 256);

    matmul_forward<<<blocks, threads>>>(x, w, y);
    hipDeviceSynchronize();

    for (int i = 0; i < N * D; ++i) grad_out[i] = 1.0f;

    matmul_backward<<<(D + 255) / 256, threads>>>(grad_out, x, grad_w);
    hipDeviceSynchronize();

    sgd_update<<<(D * D + 255) / 256, threads>>>(w, grad_w, lr);
    hipDeviceSynchronize();

    std::cout << "One training iteration complete." << std::endl;

    hipFree(x); hipFree(w); hipFree(y); hipFree(grad_out); hipFree(grad_w);
    return 0;
}
